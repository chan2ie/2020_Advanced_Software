#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#define BLOCK_SIZE 88

#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {

	const unsigned block_id = blockIdx.x;
	const unsigned thread_id = threadIdx.x;
	const unsigned r = block_id * BLOCK_SIZE + thread_id;
	
	int temp;
	int w = 2;

	for (int c = 0; c < width; c++) {

		temp = 0;
		for (int k = -w; k <= w; k++) {
			for (int l = -w; l <= w; l++) {
				if (r + k < 0 || r + k >= height || c + l < 0 || c + l >= width) {
					continue;
				}
				else {
					temp += d_bitmaps[(r + k) * width + c + l] * constant_gaussian_kernel[(k + w) * 5 + l + w];
				}
			}
		}

		d_Gaussian[r * width + c] = temp;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	int temp;
	int w = 2;

	const unsigned block_id = blockIdx.x;
	const unsigned thread_id = threadIdx.x;
	const unsigned r = block_id * BLOCK_SIZE + thread_id;

	if (thread_id == 0) {
		for (int i = -w; i < 0; i++) {
			for (int j = 0; j < width; j++) {
				if (r + i < 0) sharedBuffer[(i + w) * width + j] = 0;
				else if(r + i <= height) sharedBuffer[(i + w) * width + j] = d_bitmaps[(r + i) * width + j];
			}
		}
	}
	if (thread_id == BLOCK_SIZE - 1) {
		for (int i = 1; i <= w; i++) {
			for (int j = 0; j < width; j++) {
				if (r + i >= height) sharedBuffer[(thread_id + i + w) * width + j] = 0;
				else sharedBuffer[(thread_id + i + w) * width + j] = d_bitmaps[(r + i) * width + j];
			}
		}
	}

	for (int j = 0; j < width; j++) {
		sharedBuffer[(thread_id + w) * width + j] = d_bitmaps[r * width + j];
	}
	__syncthreads();

	for (int c = 0; c < width; c++) {

		temp = 0;
		for (int k = -w; k <= w; k++) {
			for (int l = -w; l <= w; l++) {
				if ( c + l < 0 || c + l >= width) {
					continue;
				}
				else {
					temp += sharedBuffer[(thread_id + w+ k) * width + c + l] * constant_gaussian_kernel[(k + w) * 5 + l + w];
				}
			}
		}

		d_Gaussian[r * width + c] = temp;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{

	Set_Gaussain_Kernel();
	/*Todo*/
	hipError_t cudaStatus;

	CUDA_CALL(hipSetDevice(0));

	unsigned char* d_bitmaps, * d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));
	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	dim3 blockDim(BLOCK_SIZE);
	dim3 gridDim(height/BLOCK_SIZE);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDim >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << <gridDim, blockDim, sizeof(unsigned char) * (BLOCK_SIZE + 2 * 2) * width >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));

	hipFree(d_Gaussian);
	hipFree(d_bitmaps);

	return device_time;
}